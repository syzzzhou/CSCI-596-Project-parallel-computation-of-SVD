#include <stdio.h>
#include <math.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#define ROW 6  // Example row size
#define COL 3  // Example column size
#define ITERATION 100
#define THRESHOLD 1e-6
#define NUM_BLOCK 5  // Number of thread blocks
#define NUM_THREAD ((ROW + NUM_BLOCK - 1) / NUM_BLOCK)  // Threads per block, dividing rows among blocks

// CUDA kernel for orthogonalization
__global__ void orthogonal_kernel(double *matrix, double *V, bool *pass, int row, int col, int c1, int c2) {
    int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (globalIdx >= row) return;

    __shared__ double Ci[ROW];
    __shared__ double Cj[ROW];

    // Load columns into shared memory
    if (globalIdx < row) {
        Ci[globalIdx] = matrix[globalIdx * col + c1];
        Cj[globalIdx] = matrix[globalIdx * col + c2];
    }
    __syncthreads();

    // Calculate inner product and lengths
    double inner_prod = 0, len1 = 0, len2 = 0;
    for (int i = 0; i < row; ++i) {
        inner_prod += Ci[i] * Cj[i];
        len1 += Ci[i] * Ci[i];
        len2 += Cj[i] * Cj[i];
    }

    if (fabs(inner_prod) < THRESHOLD) {
        if (globalIdx == 0) *pass = true;
        return;
    }
    if (globalIdx == 0) *pass = false;

    if (len1 < len2) {
        for (int i = 0; i < row; ++i) {
            double temp = Ci[i];
            Ci[i] = Cj[i];
            Cj[i] = temp;
        }
    }

    double tao = (len1 - len2) / (2 * inner_prod);
    double tan = (tao > 0 ? 1 : -1) / (fabs(tao) + sqrt(1 + tao * tao));
    double cos = 1 / sqrt(1 + tan * tan);
    double sin = cos * tan;

    for (int i = 0; i < row; ++i) {
        double var1 = Ci[i] * cos + Cj[i] * sin;
        double var2 = Cj[i] * cos - Ci[i] * sin;
        Ci[i] = var1;
        Cj[i] = var2;
    }

    for (int i = 0; i < row; ++i) {
        matrix[i * col + c1] = Ci[i];
        matrix[i * col + c2] = Cj[i];
    }
}

int main(int argc, char **argv) {
    double A[ROW][COL] = {
        {6, 5, 1},
        {9, 8, 4},
        {8, 5, 2},
        {4, 6, 9},
        {1, 2, 3},
        {2, 1, 4}};
    double V[COL][COL] = {0};
    double S[ROW][COL] = {0};
    double U[ROW][ROW] = {0};
    bool pass;

    for (int i = 0; i < COL; ++i) {
        V[i][i] = 1.0;
    }

    double *d_A, *d_V;
    bool *d_pass;
    size_t matrix_size = ROW * COL * sizeof(double);
    size_t vector_size = COL * COL * sizeof(double);

    // Allocate device memory
    hipMalloc((void **)&d_A, matrix_size);
    hipMalloc((void **)&d_V, vector_size);
    hipMalloc((void **)&d_pass, sizeof(bool));

    // Copy data to device
    hipMemcpy(d_A, A, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, vector_size, hipMemcpyHostToDevice);

    dim3 dimBlock(NUM_THREAD);
    dim3 dimGrid(NUM_BLOCK);

    for (int iter = 0; iter < ITERATION; ++iter) {
        pass = true;
        hipMemcpy(d_pass, &pass, sizeof(bool), hipMemcpyHostToDevice);

        for (int i = 0; i < COL; ++i) {
            for (int j = i + 1; j < COL; ++j) {
                orthogonal_kernel<<<dimGrid, dimBlock>>>(d_A, d_V, d_pass, ROW, COL, i, j);
                hipDeviceSynchronize();
            }
        }

        hipMemcpy(&pass, d_pass, sizeof(bool), hipMemcpyDeviceToHost);
        if (pass) break;
    }

    // Copy results back to host
    hipMemcpy(A, d_A, matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(V, d_V, vector_size, hipMemcpyDeviceToHost);

    // Calculate singular values (S matrix)
    double E[COL] = {0};
    int nonzero = 0;
    for (int i = 0; i < COL; ++i) {
        double norm = 0;
        for (int j = 0; j < ROW; ++j) {
            norm += A[j][i] * A[j][i];
        }
        if (norm > THRESHOLD) nonzero++;
        E[i] = sqrt(norm);
    }

    for (int i = 0; i < ROW; ++i) {
        for (int j = 0; j < COL; ++j) {
            S[i][j] = (i == j) ? E[i] : 0.0;
        }
    }

    // Calculate U matrix
    for (int i = 0; i < ROW; ++i) {
        for (int j = 0; j < nonzero; ++j) {
            U[i][j] = A[i][j] / E[j];
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_V);
    hipFree(d_pass);

    // Print results
    printf("Matrix A (after orthogonalization):\n");
    for (int i = 0; i < ROW; ++i) {
        for (int j = 0; j < COL; ++j) {
            printf("%f ", A[i][j]);
        }
        printf("\n");
    }

    printf("Matrix S:\n");
    for (int i = 0; i < ROW; ++i) {
        for (int j = 0; j < COL; ++j) {
            printf("%f ", S[i][j]);
        }
        printf("\n");
    }

    printf("Matrix V:\n");
    for (int i = 0; i < COL; ++i) {
        for (int j = 0; j < COL; ++j) {
            printf("%f ", V[i][j]);
        }
        printf("\n");
    }

    printf("Matrix U:\n");
    for (int i = 0; i < ROW; ++i) {
        for (int j = 0; j < ROW; ++j) {
            printf("%f ", U[i][j]);
        }
        printf("\n");
    }

    return 0;
}
